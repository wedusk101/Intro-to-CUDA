
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>



__global__ void init_cuda_rand(hiprandState *stateX, hiprandState *stateY, unsigned long seed)
{
	int idx = threadIdx.x;
	hiprand_init(seed, idx, 0, stateX);
	hiprand_init(seed, idx, 0, stateY);
}

__global__ void getRandom(hiprandState *globalStateX, hiprandState *globalStateY, float *d_randArrayX, float *d_randArrayY)
{
	int idx = threadIdx.x;
	hiprandState localStateX = globalStateX[idx];
	hiprandState localStateY = globalStateY[idx];
	float randValX = hiprand_uniform(&localStateX);
	float randValY = hiprand_uniform(&localStateY);
	d_randArrayX[idx] = randValX;
	d_randArrayY[idx] = randValY;
	globalStateX[idx] = localStateX;
	globalStateY[idx] = localStateY;
}

__global__ void calcPi(int *d_sampleArray, float *d_randArrayX, float *d_randArrayY, int samples)
{
	int idx = threadIdx.x;
	float x = 0, y = 0;
	int count = 0;
	for (int i = 0; i < samples * samples; i++)
	{
		x = d_randArrayX[idx];
		y = d_randArrayY[idx];
		if (x * x + y * y <= 1)
			count++;
	}
	d_sampleArray[idx] = count;
}

int main()
{
	const int ARRAY_SIZE = 128;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	int h_samples[ARRAY_SIZE];
	int samples = 1000;
	
	hiprandState *d_randStatesX;
	hiprandState *d_randStatesY;
	float *d_randArrayX;
	float *d_randArrayY;
	int *d_sampleArray;

	hipMalloc((void**)&d_randArrayX, ARRAY_SIZE * sizeof(float));
	hipMalloc((void**)&d_randArrayY, ARRAY_SIZE * sizeof(float));
	hipMalloc((void**)&d_randStatesX, ARRAY_SIZE * sizeof(hiprandState));
	hipMalloc((void**)&d_randStatesY, ARRAY_SIZE * sizeof(hiprandState));
	hipMalloc((void**)&d_sampleArray, ARRAY_BYTES);

	init_cuda_rand <<<1, ARRAY_SIZE>>> (d_randStatesX, d_randStatesY, time(NULL));
	getRandom <<<1, ARRAY_SIZE>>> (d_randStatesX, d_randStatesY, d_randArrayX, d_randArrayY);
	calcPi <<<1, ARRAY_SIZE>>> (d_sampleArray, d_randArrayX, d_randArrayY, samples);

	hipMemcpy(h_samples, d_sampleArray, ARRAY_BYTES, hipMemcpyDeviceToHost);

	long s = 0;
	for (int i = 0; i < ARRAY_SIZE; i++)
		s += h_samples[i];

	hipFree(d_randArrayX);
	hipFree(d_randArrayY);
	hipFree(d_randStatesX);
	hipFree(d_randStatesY);
	hipFree(d_sampleArray);

	std::cout << (4 * s) / (double)(ARRAY_SIZE * samples * samples) << std::endl;
}