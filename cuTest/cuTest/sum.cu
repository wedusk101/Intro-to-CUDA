#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>



__global__ void square(int *d_in, int *d_out)
{
	int idx = threadIdx.x;
	int x = d_in[idx];
	d_out[idx] = x * x;
}

int main()
{
	const int ARRAY_SIZE = 128;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	int h_in[ARRAY_SIZE];
	int h_out[ARRAY_SIZE];
	int *d_in;
	int *d_out;

	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	for (int i = 0; i < ARRAY_SIZE; i++)
		h_in[i] = i;

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	square <<<1, ARRAY_SIZE >>> (d_in, d_out);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	for (int i = 0; i < ARRAY_SIZE; i++)
		std::cout << h_out[i] << std::endl;
}