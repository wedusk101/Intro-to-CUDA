#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <fstream>
#include <cmath>
#include <iostream> 
#include <string>
#include <chrono>
#include <climits>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cstring>


void errorCheck(hipError_t code, const char *func, const char *fileName, const int line)
{
	if (code)
	{
		std::cerr << "CUDA error = " << (int)code << " in file: " <<
			fileName << " function: " << func << " on line: " << line << "\n";
		hipDeviceReset();
		exit(1);
	}
}

#define cudaErrorCheck(arg) errorCheck( (arg), #arg, __FILE__, __LINE__ )

struct Vec3
{
	float x;
	float y;
	float z;

	__host__ __device__ Vec3() : x(0), y(0), z(0) {}
	__host__ __device__ Vec3(const float &x_, const float &y_, const float &z_) : x(x_), y(y_), z(z_) {}

	__host__ __device__ float getMagnitude() const
	{
		return sqrt(x * x + y * y + z * z);
	}

	__host__ __device__ Vec3 getNormalized() const
	{
		float mag = getMagnitude();
		return Vec3(x / mag, y / mag, z / mag);
	}

	__host__ __device__ Vec3 operator+(const Vec3 &v) const // addition
	{
		return Vec3(x + v.x, y + v.y, z + v.z);
	}

	__host__ __device__ Vec3 operator-(const Vec3 &v) const // subtraction
	{
		return Vec3(x - v.x, y - v.y, z - v.z);
	}

	__host__ __device__ Vec3 operator*(const float &c) const // scalar multiplication
	{
		return Vec3(c * x, c * y, c * z);
	}

	__host__ __device__ Vec3 operator/(const float &c) const // scalar division
	{
		return Vec3(x / c, y / c, z / c);
	}

	__host__ __device__ Vec3& operator+=(const Vec3 &v) // addition
	{
		x += v.x;
		y += v.y;
		z += v.z;
		return *this;
	}

	__host__ __device__ Vec3& operator/=(const float &c) // scalar division
	{
		x /= c;
		y /= c;
		z /= c;
		return *this;
	}

	__host__ __device__ float operator%(const Vec3 &v) const // dot product
	{
		return x * v.x + y * v.y + z * v.z;
	}

	__host__ __device__ Vec3 operator&(const Vec3 &v) const // cross product
	{
		return Vec3(y * v.z - v.y * z, z * v.x - x * v.z, x * v.y - y * v.x);
	}

	__host__ __device__ float dot(const Vec3 &v) const // dot product
	{
		return x * v.x + y * v.y + z * v.z;
	}
};

struct Ray
{
	Vec3 o; // origin
	Vec3 d; // direction
	mutable float t;
	float tMin;
	mutable float tMax;

	__host__ __device__ Ray(const Vec3 &o_, const Vec3 &d_) : o(o_), d(d_), t(INT_MAX), tMin(0.1), tMax(INT_MAX) {}
};

struct Geometry
{
	Vec3 color;

	__host__ __device__ virtual bool intersects(const Ray &ray) const = 0;
	__host__ __device__ virtual Vec3 getNormal(const Vec3 &point) const = 0;
};

struct Sphere : public Geometry
{
	Vec3 center;
	float radius;

	__host__ __device__ Sphere(const Vec3 &c, const float &rad, const Vec3 &col) : center(c), radius(rad)
	{
		color = col;
	}

	__host__ __device__ Vec3 getNormal(const Vec3 &point) const // returns the surface normal at a point
	{
		return (point - center) / radius;
	}

	__host__ __device__ bool intersects(const Ray &ray) const
	{
		const float eps = 1e-4;
		const Vec3 oc = ray.o - center;
		const float b = 2 * (ray.d % oc);
		const float a = ray.d % ray.d;
		const float c = (oc % oc) - (radius * radius);
		float delta = b * b - 4 * a * c;
		if (delta < eps) // discriminant is less than zero
			return false;
		delta = sqrt(delta);
		const float t0 = (-b + delta) / (2 * a);
		const float t1 = (-b - delta) / (2 * a);
		ray.t = (t0 < t1) ? t0 : t1;
		if (ray.t >= ray.tMin && ray.t <= ray.tMax)
		{
			ray.tMax = ray.t;
			return true;
		}
		else
			return false;
	}
};

struct Plane : public Geometry
{
	Vec3 normal; // normal of the plane
	Vec3 point; // a point on the plane


	__host__ __device__ Plane(const Vec3 &n, const Vec3 &p, const Vec3 &col) : normal(n), point(p)
	{
		color = col;
	}

	__host__ __device__ Vec3 getNormal(const Vec3 &point) const
	{
		return normal;
	}

	__host__ __device__ bool intersects(const Ray &ray) const
	{
		const double eps = 1e-4;
		double parameter = ray.d % normal;
		if (fabs(parameter) < eps) // ray is parallel to the plane
			return false;
		ray.t = ((point - ray.o) % normal) / parameter;
		if (ray.t >= ray.tMin && ray.t <= ray.tMax)
		{
			ray.tMax = ray.t;
			return true;
		}
		else
			return false;
	}
};

struct Light
{
	Vec3 position;
	float radius;
	Vec3 color;
	float intensity;

	__host__ __device__ Light(const Vec3 &position_, const float &radius_, const Vec3 &color_, const float &intensity_) : position(position_), radius(radius_), color(color_), intensity(intensity_) {}
};

struct Camera
{
	Vec3 position;
	Vec3 direction;

	// add a lower left corner for orientation

	__host__ __device__ Camera(const Vec3 &pos, const Vec3 &dir) : position(pos), direction(dir) {}

	__host__ __device__ Ray getRay(int x, int y, float rand) const
	{
		double offsetX = (float)x + rand;
		double offsetY = (float)y + rand;
		return Ray(Vec3(offsetX, offsetY, 0), direction);
	}
};

__device__ Vec3 colorModulate(const Vec3 &lightColor, const Vec3 &objectColor) // performs component wise multiplication for colors  
{
	return Vec3(lightColor.x * objectColor.x, lightColor.y * objectColor.y, lightColor.z * objectColor.z);
}

__device__ void clamp(Vec3 &col)
{
	col.x = (col.x > 1) ? 1 : (col.x < 0) ? 0 : col.x;
	col.y = (col.y > 1) ? 1 : (col.y < 0) ? 0 : col.y;
	col.z = (col.z > 1) ? 1 : (col.z < 0) ? 0 : col.z;
}

__device__ Vec3 getPixelColor(Ray &cameraRay, Geometry **scene, int sceneSize, const Light *light)
{
	Vec3 pixelColor;
	Vec3 white(1, 1, 1);
	bool hitStatus = false;
	int hitIndex = 0;
	for (int i = 0; i < sceneSize; ++i)
	{
		if (scene[i]->intersects(cameraRay))
		{
			hitStatus = true;		
			hitIndex = i;
		}
	}

	if (hitStatus)
	{
		Vec3 surf = cameraRay.o + cameraRay.d * cameraRay.tMax; // point of intersection
		Vec3 L = (light->position - surf).getNormalized();


		// check for shadows
		Ray shadowRay(surf, L);
		for (int i = 0; i < sceneSize; ++i)
			if (scene[i]->intersects(shadowRay))
				return pixelColor;

		Vec3 N = scene[hitIndex]->getNormal(surf).getNormalized();
		float diffuse = L.dot(N);
		pixelColor = (colorModulate(light->color, scene[hitIndex]->color) + white * diffuse) * light->intensity;
	}
	return pixelColor;
}

__global__ void render(Vec3 *fb, int width, int height, int spp, const Camera *camera, Geometry **scene, int sceneSize, const Light *light, hiprandState *globalRandState)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int idx = threadIdx.x;
	Vec3 pixelColor;
	int index = y * width + x;
	for (int i = 0; i < spp; i++)
	{
		hiprandState localState = globalRandState[idx];
		float r = hiprand_uniform(&localState);
		globalRandState[idx] = localState;
		if ((x >= width) || (y >= height))
			return;		
		Ray cameraRay = camera->getRay(x, y, r);
		pixelColor += getPixelColor(cameraRay, scene, sceneSize, light);		
	}
	pixelColor /= (float)spp;
	clamp(pixelColor);
	fb[index] = pixelColor;
}

__global__ void initScene(int width, int height, Camera *camera, Geometry **scene, Light *light)
{
	Vec3 white(1, 1, 1);
	Vec3 blue(0, 0, 1);
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		// light = new Light(Vec3(0.8 * width, 0.25 * height, 100), 1, white, 0.5);
		// camera = new Camera(Vec3(0.5 * width, 0.5 * height, 0), Vec3(0, 0, 1));

		light->position = Vec3(0.8 * width, 0.25 * height, 100);
		light->radius = 1;
		light->color = Vec3(1, 1, 1);
		light->intensity = 0.5;

		camera->position = Vec3(0.5 * width, 0.5 * height, 0);
		camera->direction = Vec3(0, 0, 1);

		scene[0] = new Sphere(Vec3(0.5 * width, 0.45 * height, 1000), 100, Vec3(1, 0, 0));
		scene[1] = new Sphere(Vec3(0.65 * width, 0.2 * height, 600), 50, Vec3(0, 0, 1));
		scene[2] = new Plane(Vec3(0, 0, -1), Vec3(0.5 * width, 0.5 * height, 1500), Vec3(1, 1, 0));
		scene[3] = new Sphere(Vec3(0.5 * width, 0.52 * height, 700), 35, Vec3(0, 1, 1));
	}
}

__global__ void initRandState(hiprandState *randState, unsigned long seed)
{
	int idx = threadIdx.x;
	hiprand_init(seed, idx, 0, &randState[idx]);
}

int isNumber(char *input)
{
	int len = strlen(input);
	for (int i = 0; i < len; ++i)
		if (!isdigit(input[i]))
			return 0;

	return 1;
}

int main(int argc, char* argv[])
{
	int width = 2560;
	int height = 1440;
	int spp = 8;
	int tx = 8;
	int ty = 8;
	int nThreads = 1024;

	// setup multithreading and benchmark parameters

	int nBenchLoops = 1;
	bool isBenchmark = false;

	for (int i = 0; i < argc; i++) // process command line args
	{
		if (!strcmp(argv[i], "-bench")) // usage -bench numberLoops
		{
			isBenchmark = true;
			if (i + 1 < argc)
			{
				if (isNumber(argv[i + 1]))
					nBenchLoops = atoi(argv[i + 1]); // number of times to loop in benchmark mode
				else
				{
					std::cout << "Invalid benchmark loop count provided. Using default value.\n";
					nBenchLoops = 5;
				}
			}
			else
			{
				std::cout << "Benchmark loop count not provided. Using default value.\n";
				nBenchLoops = 5;
			}
		}

		if (!strcmp(argv[i], "-width"))
		{
			if (i + 1 < argc)
			{
				if (isNumber(argv[i + 1]))
					width = atoi(argv[i + 1]); 
				else
					std::cout << "Invalid image width provided. Using default value.\n";
			}
			else
				std::cout << "Image width not provided. Using default value.\n";
		}

		if (!strcmp(argv[i], "-height"))
		{
			if (i + 1 < argc)
			{
				if (isNumber(argv[i + 1]))
					height = atoi(argv[i + 1]);
				else
					std::cout << "Invalid image height provided. Using default value.\n";
			}
			else
				std::cout << "Image height not provided. Using default value.\n";
		}

		if (!strcmp(argv[i], "-spp"))
		{
			if (i + 1 < argc)
			{
				if (isNumber(argv[i + 1]))
					spp = atoi(argv[i + 1]);
				else
					std::cout << "Invalid sample count provided. Using default value.\n";
			}
			else
				std::cout << "Sample count not provided. Using default value.\n";
		}
	}

	if (argc == 1)
		std::cout << "Arguments not provided. Using default values.\n";

	// colors (R, G, B)
	const Vec3 white(1, 1, 1);
	const Vec3 black(0, 0, 0);
	const Vec3 red(1, 0, 0);
	const Vec3 green(0, 1, 0);
	const Vec3 blue(0, 0, 1);
	const Vec3 cyan(0, 1, 1);
	const Vec3 magenta(1, 0, 1);
	const Vec3 yellow(1, 1, 0);

	Light *light; 
	Camera *camera;
	Geometry **scene;
	int sceneSize = 4;
	
	int numPixels = width * height;
	size_t fbSize = numPixels * sizeof(Vec3);

	Vec3 *fb;
	hiprandState *d_randState;

	cudaErrorCheck(hipMallocManaged((void**)&d_randState, nThreads * sizeof(hiprandState)));
	cudaErrorCheck(hipMallocManaged((void**)&fb, fbSize));
	cudaErrorCheck(hipMallocManaged((void**)&light, sizeof(Light)));
	cudaErrorCheck(hipMallocManaged((void**)&camera, sizeof(Camera)));
	cudaErrorCheck(hipMallocManaged((void**)&scene, sizeof(Geometry*)));
	
	dim3 threadsPerBlock(tx, ty);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

	auto start = std::chrono::high_resolution_clock::now();

	if (isBenchmark)
		std::cout << "\nRunning in benchmark mode. Looping " << nBenchLoops << " times.\n";
	std::cout << "\nRendering...\n";

	initRandState << <1, nThreads >> > (d_randState, time(NULL));
	initScene << <1, 1 >> > (width, height, camera, scene, light);
	for (int run = 0; run < nBenchLoops; run++)
	{
		render << <numBlocks, threadsPerBlock >> > (fb, width, height, spp, camera, scene, sceneSize, light, d_randState);
		cudaErrorCheck(hipGetLastError());
		cudaErrorCheck(hipDeviceSynchronize());
	}

	auto stop = std::chrono::high_resolution_clock::now();

	std::ofstream out("result.ppm"); // creates a PPM image file for saving the rendered output
	out << "P3\n" << width << " " << height << "\n255\n";

	for (int i = 0; i < numPixels; ++i)
		out << (int)(255.99 * fb[i].x) << " " << (int)(255.99 * fb[i].y) << " " << (int)(255.99 * fb[i].z) << "\n"; // write out the pixel values

	std::cout << "\nTime taken was " << (std::chrono::duration_cast<std::chrono::milliseconds>(stop - start)).count() << " milliseconds." << std::endl;
	cudaErrorCheck(hipFree(fb));
	cudaErrorCheck(hipFree(light));
	cudaErrorCheck(hipFree(camera));
	cudaErrorCheck(hipFree(scene));
}