
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>
#include <chrono>
#include <iomanip>
#include <cstdint>


const int nThreads = 1024;

void errorCheck(hipError_t code, const char *func, const char *fileName, const int line)
{
	if (code)
	{
		std::cerr << "CUDA error = " << (int)code << " in file: " <<
			fileName << " function: " << func << " on line: " << line << "\n";
		hipDeviceReset();
		exit(1);
	}
}

#define cudaErrorCheck(arg) errorCheck( (arg), #arg, __FILE__, __LINE__ )


__global__ void initRandState(hiprandState *randState, unsigned long seed)
{
	int idx = threadIdx.x;
	hiprand_init(seed, idx, 0, &randState[idx]);
}

__global__ void calcPi(hiprandState *globalRandState, size_t *d_sampleArray, size_t samples)
{
	int idx = threadIdx.x;
	float x = 0, y = 0;
	size_t count = 0;
	size_t max = (samples * samples) / nThreads;
	hiprandState localState = globalRandState[idx];
	for (size_t i = 0; i < max; ++i)
	{
		x = hiprand_uniform(&localState);
		y = hiprand_uniform(&localState);
		if (x * x + y * y <= 1)
			count++;
	}
	globalRandState[idx] = localState;
	d_sampleArray[idx] = count;
}

int main()
{
	size_t *h_samples = new size_t[nThreads];
	size_t samples;

	std::cout << "No. of samples: " << std::endl;
	std::cin >> samples;	
	
	hiprandState *d_randState;
	size_t *d_sampleArray;

	cudaErrorCheck(hipMalloc((void**)&d_randState, nThreads * sizeof(hiprandState)));
	cudaErrorCheck(hipMalloc((void**)&d_sampleArray, nThreads * sizeof(size_t)));

	auto start = std::chrono::high_resolution_clock::now();
	initRandState <<<1, nThreads>>> (d_randState, time(NULL));
	calcPi <<<1, nThreads>>> (d_randState, d_sampleArray, samples);

	cudaErrorCheck(hipDeviceSynchronize());
	cudaErrorCheck(hipMemcpy(h_samples, d_sampleArray, nThreads * sizeof(size_t), hipMemcpyDeviceToHost));

	size_t s = 0;
	for (int i = 0; i < nThreads; i++)
		s += h_samples[i];

	long double pi = (4 * s) / (long double)(samples * samples);
	auto stop = std::chrono::high_resolution_clock::now();
	auto diff = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
	

	std::cout << std::setprecision(12) << "Approx. value of Pi is: " << pi << std::endl;
	std::cout << "\nTime taken is " << diff.count() << " milliseconds." << std::endl;

	cudaErrorCheck(hipFree(d_randState));
	cudaErrorCheck(hipFree(d_sampleArray));
	delete[] h_samples;
}